#include "hip/hip_runtime.h"
// example1.cpp : Defines the entry point for the console application.
//
#include <stdio.h>
#include <limits.h>

#include "scenes.cuh"
#include "defs.cuh"

#define MAX_DEPTH 10
 

    
__global__ void render_block(Scene *scene, RenderSettings settings, float3 *a, int N, int startidx, int samples) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        float y = ((idx + startidx) / settings.width);
        float x = ((idx + startidx) - y * settings.width) / (float) settings.width;
        y /= (float) settings.height;
        a[idx].x = a[idx].y = a[idx].z = 0.0f;
        PathVertex path[MAX_DEPTH];
        SurfacePoint intersectP;
        Ray world_ray;
        for (unsigned i = 0; i < samples; ++i) {
            calc_world_ray(scene, x + urand(threadIdx.x) / (float) settings.width,
                    y + urand(threadIdx.x) / (float) settings.height, &world_ray);

            if (calc_nearest_intersection(scene, &world_ray, &intersectP)) {

                int num = generate_path(scene, &world_ray, threadIdx.x, &path[0], MAX_DEPTH);
                evaluate_path(scene, threadIdx.x, &path[0], num, &a[idx]);
                
                //                calc_direct_illumination(&intersectP, scene,
                //                        urand(threadIdx.x), urand(threadIdx.x), &a[idx]);
                
      //a[idx].x = a[idx].z = 1.0f;
   // a[idx].x = a[idx].y = 0.0f;
            }
        }

        vec3_scalar_mult(&a[idx], 1.0f / (float) samples, &a[idx]);
  
    }
   
}


void setGPU(int gpuNo) {
	CUDA_SAFE_CALL(hipSetDevice(gpuNo + 1));
}

void render(const RenderSettings *settings, float3 *host_image, ROI roi, int gpuNo) {
  /* unsigned z =roi.startIdx;
   for(;z < roi.startIdx + roi.length;++z) {
   		host_image[z].x = host_image[z].y = 1.0f;
   		
   }*/
   // memset(&host_image[roi.startIdx], 10.0f, sizeof(float3) * roi.length);
//return;
    float3 *device_image, *host_image_tmp;

    Scene *scene = generate_diffuse_spheres(*settings);

    int N = 900000 / settings->samples; //avoid the lame timeout
    if(N > roi.length) N = roi.length; 
//    const size_t host_size = settings->width * settings->height * sizeof (float3);
    const size_t host_size = roi.length * sizeof (float3);
    const size_t device_size = sizeof (float3) * N; //can't allocate too much at once

    Scene *device_scene;
   
    const int num_iterations = host_size / device_size; //really will need to do one more iteration for remainder
    int block_size = 256;
    int n_blocks = N / block_size + (N % block_size == 0 ? 0 : 1);

    //allocate on host & device
   
    hipEvent_t ev_1, ev_2;
    hipEventCreate(&ev_1);
    hipEventCreate(&ev_2);
    hipEventRecord(ev_1, 0);
   
    host_image_tmp = (float3 *) malloc(device_size);
    hipMalloc((void **) & device_image, device_size);
    hipMalloc((void **) & device_scene, sizeof (Scene));
    CUDA_SAFE_CALL(hipMemcpy(device_scene, scene, sizeof (Scene), hipMemcpyHostToDevice));

    unsigned i = 0;
    for (; i <= num_iterations; ++i) {
        CUDA_SAFE_CALL(hipMemcpy(device_image, host_image_tmp, device_size, hipMemcpyHostToDevice));
        render_block <<< n_blocks, block_size >>> (device_scene, *settings, device_image, N, N*i+ roi.startIdx , settings->samples); //oh snap
        hipDeviceSynchronize();
        CUDA_SAFE_CALL(hipMemcpy(host_image_tmp, device_image, device_size, hipMemcpyDeviceToHost));
        if(i == num_iterations) { //if were on the last block 
            memcpy(&host_image[N * i + roi.startIdx], host_image_tmp, sizeof(float3) * (roi.length % N));
        }
        else {
            memcpy(&host_image[N * i + roi.startIdx], host_image_tmp, device_size);
        }
    }
    hipEventRecord(ev_2, 0);
    hipEventQuery(ev_1);
    hipEventQuery(ev_2);
    hipEventSynchronize(ev_1);
    hipEventSynchronize(ev_2);
    float time_e;
    hipEventElapsedTime(&time_e, ev_1, ev_2);
    printf("\033[01mGPU %d \t \033[0mTime taken: %f ms", gpuNo, time_e);
    printf(", Last error: %s\n", hipGetErrorString(hipGetLastError()));

    free(host_image_tmp);
    hipFree(device_image);
//    CUDA_SAFE_CALL(hipFree(&device_scene->mObjects));
//    CUDA_SAFE_CALL(hipFree(&device_scene->mEmitters));
    hipFree(device_scene);
    
    releaseScene(scene);
   // hipDeviceReset();
}



void DisplayProperties(hipDeviceProp_t* pDeviceProp) {
    if (!pDeviceProp) return;
    printf("\n\033[00mDevice Name\t\t\t\t - %s \033[0m ", pDeviceProp->name);
   /* printf("\n--------------------------------------------------------------------------");
    printf("\nTotal Global Memory\t\t\t - %d KB", (int)pDeviceProp->totalGlobalMem / 1024);
    printf("\nShared memory available per block \t - %d KB", (int)pDeviceProp->sharedMemPerBlock / 1024);
    printf("\nNumber of registers per thread block \t - %d", pDeviceProp->regsPerBlock);
    printf("\nWarp size in threads \t\t\t - %d", pDeviceProp->warpSize);
    printf("\nMemory Pitch \t\t\t\t - %d bytes", (int)pDeviceProp->memPitch);
    printf("\nMaximum threads per block \t\t - %d", pDeviceProp->maxThreadsPerBlock);
    printf("\nMaximum Thread Dimension (block) \t - %d %d %d", pDeviceProp->maxThreadsDim[0], pDeviceProp->maxThreadsDim[1], pDeviceProp->maxThreadsDim[2]);
    printf("\nMaximum Thread Dimension (grid) \t - %d %d %d", pDeviceProp->maxGridSize[0], pDeviceProp->maxGridSize[1], pDeviceProp->maxGridSize[2]);
    printf("\nTotal constant memory \t\t\t - %d bytes", (int)pDeviceProp->totalConstMem);
    printf("\nCUDA ver \t\t\t\t - %d.%d", pDeviceProp->major, pDeviceProp->minor);
    printf("\nClock rate \t\t\t\t - %d KHz", pDeviceProp->clockRate);
    printf("\nTexture Alignment \t\t\t - %d bytes", (int)pDeviceProp->textureAlignment);
    printf("\nDevice Overlap \t\t\t\t - %s", pDeviceProp-> deviceOverlap ? "Allowed" : "Not Allowed");
    printf("\nNumber of Multi processors \t\t - %d\n\n", pDeviceProp->multiProcessorCount);*/
}

void print_device_properties() {
	hipDeviceProp_t deviceProp;
	int nDevCount = 0;

	hipGetDeviceCount( &nDevCount );
	printf( "\033[34;01mTotal Number of GPUs found: %d\033[0m", nDevCount );
	printf("\n--------------------------------------------------------------------------");
	for (int nDeviceIdx = 0; nDeviceIdx < nDevCount; ++nDeviceIdx )
	{
		memset( &deviceProp, 0, sizeof(deviceProp));
		if( hipSuccess == hipGetDeviceProperties(&deviceProp, nDeviceIdx))
			DisplayProperties( &deviceProp );
		else
			printf( "\n%s", hipGetErrorString(hipGetLastError()));
	}
}
